#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>

#include "../CUDA_include/cudacsr.h"
#include "../include/csr_matrix.h"
#include "../CUDA_include/csr_kernel0.cuh"
#include "../CUDA_include/csr_kernel1.cuh"
#include "../CUDA_include/csr_kernel2.cuh"
#include "../CUDA_include/csr_kernel3.cuh"
#include "../CUDA_include/csr_kernel4.cuh"
#include "../CUDA_include/csr_kernel5.cuh"

#define WARP_SIZE 32

//k5
double spmv_csr_warps_texture(CSRMatrix *csr, const double *x, double *y) {
    int M = csr->M;
    int NZ = csr->NZ;

    hipEvent_t start, stop;
    float elapsedTime;

    // Puntatori memoria GPU
    int *d_IRP, *d_JA;
    double *d_AS, *d_y;
    float *d_x;  // Texture usa float invece di double

    hipStream_t stream;
    hipStreamCreate(&stream);

    // Allocazione memoria sulla GPU
    hipMalloc(&d_IRP, (M + 1) * sizeof(int));
    hipMalloc(&d_JA, NZ * sizeof(int));
    hipMalloc(&d_AS, NZ * sizeof(double));
    hipMalloc(&d_y, M * sizeof(double));
    hipMalloc(&d_x, csr->N * sizeof(float));  // Texture usa float

    // Convertire double in float prima di copiarlo
    float *h_x = (float*)malloc(csr->N * sizeof(float));
    for (int i = 0; i < csr->N; i++) h_x[i] = (float)x[i];

    // Dati da CPU a GPU
    hipMemcpy(d_IRP, csr->IRP, (M + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_JA, csr->JA, NZ * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_AS, csr->AS, NZ * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_x, h_x, csr->N * sizeof(float), hipMemcpyHostToDevice);
    free(h_x);

    // Creazione texture object
    hipResourceDesc resDesc = {};
    resDesc.resType = hipResourceTypeLinear;
    resDesc.res.linear.devPtr = d_x;
    resDesc.res.linear.desc.f = hipChannelFormatKindFloat;
    resDesc.res.linear.desc.x = 32;
    resDesc.res.linear.sizeInBytes = csr->N * sizeof(float);

    hipTextureDesc texDesc = {};
    texDesc.readMode = hipReadModeElementType;

    hipTextureObject_t tex_x;
    hipCreateTextureObject(&tex_x, &resDesc, &texDesc, nullptr);

    // Configurazione blocchi e griglia kernel
    int warp_size = 32;
    int num_warps_per_block = 8;
    dim3 block_dim(warp_size, num_warps_per_block);
    dim3 grid_dim((M + num_warps_per_block - 1) / num_warps_per_block);

    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Avvia il cronometro
    hipEventRecord(start, stream);

    spmv_csr_warp_texture<<<grid_dim, block_dim, 0, stream>>>(M, d_IRP, d_JA, d_AS, tex_x, d_y);

    hipEventRecord(stop, stream);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    // Copia il risultato dalla GPU alla CPU
    hipMemcpyAsync(y, d_y, M * sizeof(double), hipMemcpyDeviceToHost, stream);

    // Distruggere il texture object
    hipDestroyTextureObject(tex_x);

    hipStreamSynchronize(stream);

    // Libera memoria sulla GPU
    hipFree(d_IRP);
    hipFree(d_JA);
    hipFree(d_AS);
    hipFree(d_x);
    hipFree(d_y);

    hipStreamDestroy(stream);

    return elapsedTime / 1000;  // Restituisce il tempo in secondi
}


//K4
double spmv_csr_warps_cachel2(CSRMatrix *csr, double *x, double *y) {
    int M = csr->M;
    int NZ = csr->NZ;

    hipEvent_t start, stop;
    float elapsedTime;

    // Puntatori per la memoria sulla GPU
    int *d_IRP, *d_JA;
    double *d_AS, *d_x, *d_y;

    hipStream_t stream;
    hipStreamCreate(&stream);

    // Allocazione memoria sulla GPU
    hipMalloc(&d_IRP, (M + 1) * sizeof(int));
    hipMalloc(&d_JA, NZ * sizeof(int));
    hipMalloc(&d_AS, NZ * sizeof(double));
    hipMalloc(&d_x, csr->N * sizeof(double));
    hipMalloc(&d_y, M * sizeof(double));

    // Dati da CPU a GPU
    hipMemcpyAsync(d_IRP, csr->IRP, (M + 1) * sizeof(int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_JA, csr->JA, NZ * sizeof(int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_AS, csr->AS, NZ * sizeof(double), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_x, x, csr->N * sizeof(double), hipMemcpyHostToDevice, stream);
    hipMemset(d_y, 0, M * sizeof(double));  // Inizializza d_y a zero

    // Creazione degli eventi CUDA per il timing
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Configurazione blocchi e griglia kernel
    int warps_per_block = 4;
    dim3 block_dim(32, warps_per_block); 
    dim3 grid_dim((M + warps_per_block - 1) / warps_per_block);

    // Avvia cronometro
    hipEventRecord(start, stream);

    spmv_csr_warps_cachel2<<<grid_dim, block_dim, 0, stream>>>(M, d_IRP, d_JA, d_AS, d_x, d_y);

    hipEventRecord(stop, stream);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    // Copia il risultato dalla GPU alla CPU
    hipMemcpyAsync(y, d_y, M * sizeof(double), hipMemcpyDeviceToHost, stream);

    hipStreamSynchronize(stream);

    // Libera memoria sulla GPU
    hipFree(d_IRP);
    hipFree(d_JA);
    hipFree(d_AS);
    hipFree(d_x);
    hipFree(d_y);

    // Distrugge lo stream CUDA
    hipStreamDestroy(stream);

    return elapsedTime / 1000;  // Restituisce il tempo in secondi
}





//kernel3
double spmv_csr_warps_shmem_ridpar(CSRMatrix *csr, double *x, double *y) {
    int M = csr->M;
    int NZ = csr->NZ;

    hipEvent_t start, stop;
    float elapsedTime;

    // Puntatori per la memoria sulla GPU
    int *d_IRP, *d_JA;
    double *d_AS, *d_x, *d_y;

    hipStream_t stream;
    hipStreamCreate(&stream);

    // Allocazione memoria sulla GPU
    hipMalloc(&d_IRP, (M + 1) * sizeof(int));
    hipMalloc(&d_JA, NZ * sizeof(int));
    hipMalloc(&d_AS, NZ * sizeof(double));
    hipMalloc(&d_x, csr->N * sizeof(double));
    hipMalloc(&d_y, M * sizeof(double));

    // Dati da CPU a GPU
    hipMemcpyAsync(d_IRP, csr->IRP, (M + 1) * sizeof(int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_JA, csr->JA, NZ * sizeof(int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_AS, csr->AS, NZ * sizeof(double), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_x, x, csr->N * sizeof(double), hipMemcpyHostToDevice, stream);

    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Configurazione blocchi e griglia kernel
    int warps_per_block = 4;
    dim3 block_dim(32, warps_per_block);
    dim3 grid_dim((M + warps_per_block - 1) / warps_per_block);

    size_t shared_mem_size = warps_per_block * 32 * sizeof(double);  // Allocazione della shared memory

    // Avvia cronometro
    hipEventRecord(start, stream);


    spmv_csr_warps_shmem_ridpar<<<grid_dim, block_dim, shared_mem_size>>>(M, d_IRP, d_JA, d_AS, d_x, d_y);

    hipEventRecord(stop, stream);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    // Risultato da GPU a CPU
    hipMemcpyAsync(y, d_y, M * sizeof(double), hipMemcpyDeviceToHost, stream);

    hipStreamSynchronize(stream);

    // Libera memoria sulla GPU
    hipFree(d_IRP);
    hipFree(d_JA);
    hipFree(d_AS);
    hipFree(d_x);
    hipFree(d_y);

    // Distrugge lo stream CUDA
    hipStreamDestroy(stream);

    return elapsedTime / 1000;  // Restituisce il tempo in secondi
}




//kernel2
double spmv_csr_warps_shmem(CSRMatrix *csr, double *x, double *y) {
    int M = csr->M;
    int NZ = csr->NZ;

    hipEvent_t start, stop;
    float elapsedTime;

    // Puntatori per la memoria sulla GPU
    int *d_IRP, *d_JA;
    double *d_AS, *d_x, *d_y;

    hipStream_t stream;
    hipStreamCreate(&stream);

    // Allocazione memoria sulla GPU
    hipMalloc(&d_IRP, (M + 1) * sizeof(int));
    hipMalloc(&d_JA, NZ * sizeof(int));
    hipMalloc(&d_AS, NZ * sizeof(double));
    hipMalloc(&d_x, csr->N * sizeof(double));
    hipMalloc(&d_y, M * sizeof(double));

    // Copia dati da CPU a GPU
    hipMemcpyAsync(d_IRP, csr->IRP, (M + 1) * sizeof(int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_JA, csr->JA, NZ * sizeof(int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_AS, csr->AS, NZ * sizeof(double), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_x, x, csr->N * sizeof(double), hipMemcpyHostToDevice, stream);

    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Configurazione blocchi e griglia kernel
    int warps_per_block = 4;
    dim3 block_dim(32, warps_per_block);
    dim3 grid_dim((M + warps_per_block - 1) / warps_per_block);

    // Allocazione memoria condivisa
    size_t shared_mem_size = warps_per_block * WARP_SIZE * sizeof(double);

    // Avvia cronometro
    hipEventRecord(start, stream);

    spmv_csr_warps_shmem<<<grid_dim, block_dim, shared_mem_size>>>(M, d_IRP, d_JA, d_AS, d_x, d_y);

    hipEventRecord(stop, stream);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    // Risultato da GPU a CPU
    hipMemcpyAsync(y, d_y, M * sizeof(double), hipMemcpyDeviceToHost, stream);

    hipStreamSynchronize(stream);

    // Libera memoria sulla GPU
    hipFree(d_IRP);
    hipFree(d_JA);
    hipFree(d_AS);
    hipFree(d_x);
    hipFree(d_y);

    // Distrugge lo stream CUDA
    hipStreamDestroy(stream);

    return elapsedTime / 1000;  // Restituisce il tempo in secondi
}


//kernel1
double spmv_csr_warps(CSRMatrix *h_mat, double *h_x, double *h_y) {
    int M = h_mat->M;
    int NZ = h_mat->NZ;

    hipEvent_t start, stop;
    float elapsedTime;

    // Puntatori per la memoria sulla GPU
    int *d_IRP, *d_JA;
    double *d_AS, *d_x, *d_y;

    hipStream_t stream;
    hipStreamCreate(&stream);

    // Allocazione memoria sulla GPU
    hipMalloc(&d_IRP, (M + 1) * sizeof(int));
    hipMalloc(&d_JA, NZ * sizeof(int));
    hipMalloc(&d_AS, NZ * sizeof(double));
    hipMalloc(&d_x, h_mat->N * sizeof(double));
    hipMalloc(&d_y, M * sizeof(double));

    
    // Copia dati da CPU a GPU
    hipMemcpy(d_IRP, h_mat->IRP, (M + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_JA, h_mat->JA, NZ * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_AS, h_mat->AS, NZ * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_x, h_x, h_mat->N * sizeof(double), hipMemcpyHostToDevice);
    

    // Configurazione blocchi e griglia kernel
    dim3 blockDim(WARP_SIZE, 8);
    dim3 gridDim((M + blockDim.y - 1) / blockDim.y);

    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Avvio cronometro
    hipEventRecord(start, stream);

    spmv_csr_warps<<<gridDim, blockDim, 0, stream>>>(M, d_IRP, d_JA, d_AS, d_x, d_y);

    hipEventRecord(stop, stream);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    // Risultato da GPU a CPU
    hipMemcpyAsync(h_y, d_y, M * sizeof(double), hipMemcpyDeviceToHost, stream);

    hipStreamSynchronize(stream);

    // Libera memoria sulla GPU
    hipFree(d_IRP);
    hipFree(d_JA);
    hipFree(d_AS);
    hipFree(d_x);
    hipFree(d_y);

    // Distrugge lo stream CUDA
    hipStreamDestroy(stream);

    return elapsedTime / 1000; // Converte ms in secondi
}




//kernel0
double spmv_csr_threads(CSRMatrix *h_mat, double *h_x, double *h_y) {
    int M = h_mat->M;
    int NZ = h_mat->NZ;

    // Creazione variabili per GPU
    hipEvent_t start, stop;
    float elapsedTime;
    int *d_IRP, *d_JA;
    double *d_AS, *d_x, *d_y;

    //Allocazione variabili e spostamento da host a gpu
    hipStream_t stream;
    hipStreamCreate(&stream);

    hipMalloc(&d_IRP, (M + 1) * sizeof(int));
    hipMalloc(&d_JA, NZ * sizeof(int));
    hipMalloc(&d_AS, NZ * sizeof(double));
    hipMalloc(&d_x, h_mat->N * sizeof(double));
    hipMalloc(&d_y, M * sizeof(double));

    hipMemcpyAsync(d_IRP, h_mat->IRP, (M + 1) * sizeof(int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_JA, h_mat->JA, NZ * sizeof(int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_AS, h_mat->AS, NZ * sizeof(double), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_x, h_x, h_mat->N * sizeof(double), hipMemcpyHostToDevice, stream);

    // Configurazione blocchi e griglia kernel
    int threads_per_block = 256;
    int num_blocks = (M + threads_per_block - 1) / threads_per_block;

    //Cronometro
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, stream);

    spmv_csr_threads<<<num_blocks, threads_per_block, 0, stream>>>(M, d_IRP, d_JA, d_AS, d_x, d_y);

    hipEventRecord(stop, stream);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    hipMemcpyAsync(h_y, d_y, M * sizeof(double), hipMemcpyDeviceToHost, stream);

    hipStreamSynchronize(stream);

    //Free variabili GPU
    hipFree(d_IRP);
    hipFree(d_JA);
    hipFree(d_AS);
    hipFree(d_x);
    hipFree(d_y);

    hipStreamDestroy(stream);

    return elapsedTime / 1000; //elapsedTime è nativamente in ms
}