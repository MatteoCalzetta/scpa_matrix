#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>

#include "../CUDA_include/cudacsr.h"
#include "../include/csr_matrix.h"
#include "../CUDA_include/csr_kernel0.cuh"
#include "../CUDA_include/csr_kernel1.cuh"
#include "../CUDA_include/csr_kernel2.cuh"
#include "../CUDA_include/csr_kernel3.cuh"
#include "../CUDA_include/csr_kernel4.cuh"
#include "../CUDA_include/csr_kernel5.cuh"

#define WARP_SIZE 32

#include <hip/hip_runtime.h>
#include <cstdio>


//PROVA K5
double spmv_csr_gpu_texture(CSRMatrix *csr, const double *x, double *y) {
    int M = csr->M;
    int NZ = csr->NZ;

    hipEvent_t start, stop;
    float elapsedTime;

    int *d_IRP, *d_JA;
    double *d_AS, *d_y;
    float *d_x;  // Texture usa float invece di double

    hipStream_t stream;
    hipStreamCreate(&stream);

    // Allocazione memoria sulla GPU
    hipMalloc(&d_IRP, (M + 1) * sizeof(int));
    hipMalloc(&d_JA, NZ * sizeof(int));
    hipMalloc(&d_AS, NZ * sizeof(double));
    hipMalloc(&d_y, M * sizeof(double));
    hipMalloc(&d_x, csr->N * sizeof(float));  // Texture usa float

    // Convertire double in float prima di copiarlo
    float *h_x = (float*)malloc(csr->N * sizeof(float));
    for (int i = 0; i < csr->N; i++) h_x[i] = (float)x[i];

    hipMemcpy(d_IRP, csr->IRP, (M + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_JA, csr->JA, NZ * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_AS, csr->AS, NZ * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_x, h_x, csr->N * sizeof(float), hipMemcpyHostToDevice);
    free(h_x);

    // Creazione texture object
    hipResourceDesc resDesc = {};
    resDesc.resType = hipResourceTypeLinear;
    resDesc.res.linear.devPtr = d_x;
    resDesc.res.linear.desc.f = hipChannelFormatKindFloat;
    resDesc.res.linear.desc.x = 32;
    resDesc.res.linear.sizeInBytes = csr->N * sizeof(float);

    hipTextureDesc texDesc = {};
    texDesc.readMode = hipReadModeElementType;

    hipTextureObject_t tex_x;
    hipCreateTextureObject(&tex_x, &resDesc, &texDesc, nullptr);

    // Configurazione kernel: 1 warp per riga
    int warp_size = 32;
    int num_warps_per_block = 8;  // Numero di warp per blocco
    dim3 block_dim(warp_size, num_warps_per_block);
    dim3 grid_dim((M + num_warps_per_block - 1) / num_warps_per_block);

    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Avvia il cronometro
    hipEventRecord(start, stream);

    spmv_csr_warp_texture<<<grid_dim, block_dim, 0, stream>>>(M, d_IRP, d_JA, d_AS, tex_x, d_y);

    hipEventRecord(stop, stream);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    // Copia il risultato dalla GPU alla CPU
    hipMemcpyAsync(y, d_y, M * sizeof(double), hipMemcpyDeviceToHost, stream);

    // Distruggere il texture object
    hipDestroyTextureObject(tex_x);

    // Sincronizzare lo stream prima di liberare memoria
    hipStreamSynchronize(stream);

    // Libera memoria sulla GPU
    hipFree(d_IRP);
    hipFree(d_JA);
    hipFree(d_AS);
    hipFree(d_x);
    hipFree(d_y);

    hipStreamDestroy(stream);

    return elapsedTime / 1000;  // Restituisce il tempo in secondi
}


//PROV K4
double spmv_csr_warps_shmem_ridpar_launcher(CSRMatrix *csr, double *x, double *y) {
    int M = csr->M;
    int NZ = csr->NZ;

    // Creazione eventi CUDA per misurare il tempo
    hipEvent_t start, stop;
    float elapsedTime;

    // Puntatori per la memoria sulla GPU
    int *d_IRP, *d_JA;
    double *d_AS, *d_x, *d_y;

    // Creazione di uno stream CUDA per l'asincronia
    hipStream_t stream;
    hipStreamCreate(&stream);

    // Allocazione memoria sulla GPU
    hipMalloc(&d_IRP, (M + 1) * sizeof(int));
    hipMalloc(&d_JA, NZ * sizeof(int));
    hipMalloc(&d_AS, NZ * sizeof(double));
    hipMalloc(&d_x, csr->N * sizeof(double));
    hipMalloc(&d_y, M * sizeof(double));

    // Copia dei dati dalla CPU alla GPU
    hipMemcpyAsync(d_IRP, csr->IRP, (M + 1) * sizeof(int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_JA, csr->JA, NZ * sizeof(int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_AS, csr->AS, NZ * sizeof(double), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_x, x, csr->N * sizeof(double), hipMemcpyHostToDevice, stream);
    hipMemset(d_y, 0, M * sizeof(double));  // Inizializza d_y a zero

    // Creazione degli eventi CUDA per il timing
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Configurazione del kernel
    int warps_per_block = 4;  // Numero di warps per blocco
    dim3 block_dim(32, warps_per_block);  // 32 thread per warp, più warps nel blocco
    dim3 grid_dim((M + warps_per_block - 1) / warps_per_block);  // Numero di blocchi

    // Avvia il cronometro prima dell'esecuzione del kernel
    hipEventRecord(start, stream);

    // Lancio del kernel
    spmv_csr_warps_shmem_ridpar2<<<grid_dim, block_dim, 0, stream>>>(M, d_IRP, d_JA, d_AS, d_x, d_y);

    // Ferma il cronometro
    hipEventRecord(stop, stream);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    // Copia il risultato dalla GPU alla CPU in modo asincrono
    hipMemcpyAsync(y, d_y, M * sizeof(double), hipMemcpyDeviceToHost, stream);

    // Sincronizza lo stream prima di liberare memoria
    hipStreamSynchronize(stream);

    // Libera memoria sulla GPU
    hipFree(d_IRP);
    hipFree(d_JA);
    hipFree(d_AS);
    hipFree(d_x);
    hipFree(d_y);

    // Distrugge lo stream CUDA
    hipStreamDestroy(stream);

    return elapsedTime / 1000;  // Restituisce il tempo in secondi
}





//kernel3
double spmv_csr_warps_shmem_ridpar(CSRMatrix *csr, double *x, double *y) {
    int M = csr->M;
    int NZ = csr->NZ;

    // Creazione eventi CUDA per misurare il tempo
    hipEvent_t start, stop;
    float elapsedTime;

    // Puntatori per la memoria sulla GPU
    int *d_IRP, *d_JA;
    double *d_AS, *d_x, *d_y;

    // Creazione di uno stream CUDA per l'asincronia
    hipStream_t stream;
    hipStreamCreate(&stream);

    // Allocazione memoria sulla GPU
    hipMalloc(&d_IRP, (M + 1) * sizeof(int));
    hipMalloc(&d_JA, NZ * sizeof(int));
    hipMalloc(&d_AS, NZ * sizeof(double));
    hipMalloc(&d_x, csr->N * sizeof(double));
    hipMalloc(&d_y, M * sizeof(double));

    // Copia dei dati dalla CPU alla GPU
    hipMemcpyAsync(d_IRP, csr->IRP, (M + 1) * sizeof(int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_JA, csr->JA, NZ * sizeof(int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_AS, csr->AS, NZ * sizeof(double), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_x, x, csr->N * sizeof(double), hipMemcpyHostToDevice, stream);

    // Creazione degli eventi CUDA per il timing
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int warps_per_block = 4;  // Numero di warps per blocco
    dim3 block_dim(32, warps_per_block);  // 32 thread per warp, più warps nel blocco
    dim3 grid_dim((M + warps_per_block - 1) / warps_per_block);  // Numero di blocchi

    size_t shared_mem_size = warps_per_block * 32 * sizeof(double);  // Allocazione della shared memory

    // Avvia il cronometro prima dell'esecuzione del kernel
    hipEventRecord(start, stream);

    // Lancio del kernel con shared memory
    spmv_csr_warps_shmem_ridpar<<<grid_dim, block_dim, shared_mem_size>>>(M, d_IRP, d_JA, d_AS, d_x, d_y);

    // Ferma il cronometro
    hipEventRecord(stop, stream);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    // Stampa il tempo di esecuzione
    printf("Tempo di esecuzione: %.10f ms\n", elapsedTime);

    // Copia il risultato dalla GPU alla CPU in modo asincrono
    hipMemcpyAsync(y, d_y, M * sizeof(double), hipMemcpyDeviceToHost, stream);

    // Sincronizza lo stream prima di liberare memoria
    hipStreamSynchronize(stream);

    // Libera memoria sulla GPU
    hipFree(d_IRP);
    hipFree(d_JA);
    hipFree(d_AS);
    hipFree(d_x);
    hipFree(d_y);

    // Distrugge lo stream CUDA
    hipStreamDestroy(stream);

    return elapsedTime / 1000;  // Restituisce il tempo in secondi
}




//kernel2
double spmv_csr_warps_shmem(CSRMatrix *csr, double *x, double *y) {
    int M = csr->M;
    int NZ = csr->NZ;

    // Creazione eventi CUDA per misurare il tempo
    hipEvent_t start, stop;
    float elapsedTime;

    // Puntatori per la memoria sulla GPU
    int *d_IRP, *d_JA;
    double *d_AS, *d_x, *d_y;

    // Creazione di uno stream CUDA per l'asincronia
    hipStream_t stream;
    hipStreamCreate(&stream);

    // Allocazione memoria sulla GPU
    hipMalloc(&d_IRP, (M + 1) * sizeof(int));
    hipMalloc(&d_JA, NZ * sizeof(int));
    hipMalloc(&d_AS, NZ * sizeof(double));
    hipMalloc(&d_x, csr->N * sizeof(double));
    hipMalloc(&d_y, M * sizeof(double));

    // Copia dei dati dalla CPU alla GPU
    hipMemcpyAsync(d_IRP, csr->IRP, (M + 1) * sizeof(int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_JA, csr->JA, NZ * sizeof(int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_AS, csr->AS, NZ * sizeof(double), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_x, x, csr->N * sizeof(double), hipMemcpyHostToDevice, stream);

    // Creazione degli eventi CUDA per il timing
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int warps_per_block = 4;  // Numero di warps per blocco
    dim3 block_dim(32, warps_per_block);  // 32 thread per warp, più warps nel blocco
    dim3 grid_dim((M + warps_per_block - 1) / warps_per_block);  // Numero di blocchi

    // Allocazione memoria condivisa per la riduzione
    size_t shared_mem_size = warps_per_block * WARP_SIZE * sizeof(double);

    // Avvia il cronometro prima dell'esecuzione del kernel
    hipEventRecord(start, stream);

    // Lancio del kernel
    spmv_csr_warps_shmem<<<grid_dim, block_dim, shared_mem_size>>>(M, d_IRP, d_JA, d_AS, d_x, d_y);

    // Ferma il cronometro
    hipEventRecord(stop, stream);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    // Stampa il tempo di esecuzione
    printf("Tempo di esecuzione: %.10f ms\n", elapsedTime);

    // Copia il risultato dalla GPU alla CPU in modo asincrono
    hipMemcpyAsync(y, d_y, M * sizeof(double), hipMemcpyDeviceToHost, stream);

    // Sincronizza lo stream prima di liberare memoria
    hipStreamSynchronize(stream);

    // Libera memoria sulla GPU
    hipFree(d_IRP);
    hipFree(d_JA);
    hipFree(d_AS);
    hipFree(d_x);
    hipFree(d_y);

    // Distrugge lo stream CUDA
    hipStreamDestroy(stream);

    return elapsedTime / 1000;  // Restituisce il tempo in secondi
}



// FAI VARIARE GRANDEZZA WARP PER BLOCCO E INDAGA, DIVERSE GRANDEZZE DIVERSI ERRORI NORMA
//kernel1
double spmv_csr_warps(CSRMatrix *h_mat, double *h_x, double *h_y) {
    int M = h_mat->M;
    int NZ = h_mat->NZ;

    // Creazione eventi CUDA per il timing
    hipEvent_t start, stop;
    float elapsedTime;

    // Puntatori per la memoria sulla GPU
    int *d_IRP, *d_JA;
    double *d_AS, *d_x, *d_y;

    // Creazione di uno stream CUDA per l'asincronia
    hipStream_t stream;
    hipStreamCreate(&stream);

    // Allocazione memoria sulla GPU
    hipMalloc(&d_IRP, (M + 1) * sizeof(int));
    hipMalloc(&d_JA, NZ * sizeof(int));
    hipMalloc(&d_AS, NZ * sizeof(double));
    hipMalloc(&d_x, h_mat->N * sizeof(double));
    hipMalloc(&d_y, M * sizeof(double));

    
    // Copia dati dalla CPU alla GPU in modo asincrono
    hipMemcpy(d_IRP, h_mat->IRP, (M + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_JA, h_mat->JA, NZ * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_AS, h_mat->AS, NZ * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_x, h_x, h_mat->N * sizeof(double), hipMemcpyHostToDevice);
    
    
    /*
    // Copia dati dalla CPU alla GPU in modo asincrono
    hipMemcpyAsync(d_IRP, h_mat->IRP, (M + 1) * sizeof(int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_JA, h_mat->JA, NZ * sizeof(int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_AS, h_mat->AS, NZ * sizeof(double), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_x, h_x, h_mat->N * sizeof(double), hipMemcpyHostToDevice, stream);
    */

    // Configurazione della griglia CUDA
    dim3 blockDim(WARP_SIZE, 8);  // 4 warps per blocco (128 thread)
    dim3 gridDim((M + blockDim.y - 1) / blockDim.y);

    // Creazione degli eventi CUDA per il timing
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Avvia il cronometro prima dell'esecuzione del kernel
    hipEventRecord(start, stream);

    // Lancio del kernel con lo stream
    spmv_csr_warps<<<gridDim, blockDim, 0, stream>>>(M, d_IRP, d_JA, d_AS, d_x, d_y);

    // Ferma il cronometro
    hipEventRecord(stop, stream);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    // Stampa il tempo di esecuzione
    printf("Tempo di esecuzione: %.10f ms\n", elapsedTime);

    // Copia il risultato dalla GPU alla CPU in modo asincrono
    hipMemcpyAsync(h_y, d_y, M * sizeof(double), hipMemcpyDeviceToHost, stream);

    // Sincronizza lo stream prima di liberare memoria
    hipStreamSynchronize(stream);

    // Libera memoria sulla GPU
    hipFree(d_IRP);
    hipFree(d_JA);
    hipFree(d_AS);
    hipFree(d_x);
    hipFree(d_y);

    // Distrugge lo stream CUDA
    hipStreamDestroy(stream);

    return elapsedTime / 1000; // Converte ms in secondi
}




//kernel0
double spmv_csr_threads(CSRMatrix *h_mat, double *h_x, double *h_y) {
    int M = h_mat->M;
    int NZ = h_mat->NZ;

    // Creazione variabili per GPU
    hipEvent_t start, stop;
    float elapsedTime;
    int *d_IRP, *d_JA;
    double *d_AS, *d_x, *d_y;

    //Allocazione variabili e spostamento da host a gpu
    hipStream_t stream;
    hipStreamCreate(&stream);

    hipMalloc(&d_IRP, (M + 1) * sizeof(int));
    hipMalloc(&d_JA, NZ * sizeof(int));
    hipMalloc(&d_AS, NZ * sizeof(double));
    hipMalloc(&d_x, h_mat->N * sizeof(double));
    hipMalloc(&d_y, M * sizeof(double));

    hipMemcpyAsync(d_IRP, h_mat->IRP, (M + 1) * sizeof(int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_JA, h_mat->JA, NZ * sizeof(int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_AS, h_mat->AS, NZ * sizeof(double), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_x, h_x, h_mat->N * sizeof(double), hipMemcpyHostToDevice, stream);

    // Configurazione della griglia CUDA
    int threads_per_block = 256;
    int num_blocks = (M + threads_per_block - 1) / threads_per_block;

    //Cronometro
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, stream);

    spmv_csr_threads<<<num_blocks, threads_per_block, 0, stream>>>(M, d_IRP, d_JA, d_AS, d_x, d_y);

    hipEventRecord(stop, stream);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    printf("Tempo di esecuzione: %.10f ms\n", elapsedTime);

    //Copia del risultato
    hipMemcpyAsync(h_y, d_y, M * sizeof(double), hipMemcpyDeviceToHost, stream);

    hipStreamSynchronize(stream);

    //Free variabili GPU
    hipFree(d_IRP);
    hipFree(d_JA);
    hipFree(d_AS);
    hipFree(d_x);
    hipFree(d_y);

    hipStreamDestroy(stream);

    return elapsedTime / 1000; //elapsedTime è nativamente in ms
}